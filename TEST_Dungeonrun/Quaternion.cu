#include "framework.h"
#include "framework.cuh"

hipError_t Quaternion::set_rot_matrix_CUDA() {
	hipMemcpy(d_rot_m, h_rot_m, sizeof(T_fp) * 9 * size, hipMemcpyHostToDevice);
	return hipPeekAtLastError();
}

hipError_t Quaternion::initialize_CUDA() {
	hipMalloc((void**)&d_vec.i, sizeof(T_fp) * size);
	hipMalloc((void**)&d_vec.j, sizeof(T_fp) * size);
	hipMalloc((void**)&d_vec.k, sizeof(T_fp) * size);
	hipMalloc((void**)&d_vec.w, sizeof(T_fp) * size);
	hipMalloc((void**)&d_rot_m, sizeof(T_fp) * 9 * size);
	return hipPeekAtLastError();
}

hipError_t Quaternion::_memset_CUDA() {
	hipMemcpy(d_vec.i, h_vec.i, sizeof(T_fp) * size, hipMemcpyHostToDevice);
	hipMemcpy(d_vec.i, h_vec.i, sizeof(T_fp) * size, hipMemcpyHostToDevice);
	hipMemcpy(d_vec.i, h_vec.i, sizeof(T_fp) * size, hipMemcpyHostToDevice);
	hipMemcpy(d_vec.i, h_vec.i, sizeof(T_fp) * size, hipMemcpyHostToDevice);
	return hipPeekAtLastError();
}


