#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "platform_common.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#ifndef CUDA_VECTOR_H
#include "vector.cuh"
#endif
#include "Camera.h"
#include "Trixel.h"

__global__ void init_tri_mem_cuda(Trixel::trixel_memory* t, double* point_data, u64 max_threads) {
	u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
	if (i >= max_threads) { return; }
	u64 p1 = i * 9;
	t->d_p1[i].x = point_data[p1]; 	t->d_p1[i].y = point_data[p1 + 1]; 	t->d_p1[i].z = point_data[p1 + 2];

	t->d_edges.e1.x[i] = point_data[p1 + 3] - point_data[p1];
	t->d_edges.e1.y[i] = point_data[p1 + 4] - point_data[p1 + 1];
	t->d_edges.e1.z[i] = point_data[p1 + 5] - point_data[p1 + 2];

	t->d_edges.e2.x[i] = point_data[p1 + 6] - point_data[p1];
	t->d_edges.e2.y[i] = point_data[p1 + 7] - point_data[p1 + 1];
	t->d_edges.e2.z[i] = point_data[p1 + 8] - point_data[p1 + 2];

	device_cross(&t->d_n.x[i], &t->d_n.y[i], &t->d_n.z[i], t->d_edges.e1.x[i], t->d_edges.e1.y[i], t->d_edges.e1.z[i], t->d_edges.e2.x[i], t->d_edges.e2.y[i], t->d_edges.e2.z[i]);
	device_normalize_vector(&t->d_n.x[i], &t->d_n.y[i], &t->d_n.z[i]);


}
__global__ void init_cam_tri_mem_cuda(Trixel::trixel_memory* tm, Camera::trixel_memory* cm, vector_xyz c_pos, u64 max_threads) {
	u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
	if (i >= max_threads) { return; }

	cm->d_t.x[i] = c_pos.x - tm->d_p1[i].x;	cm->d_t.y[i] = c_pos.y - tm->d_p1[i].y;	cm->d_t.z[i] = c_pos.z - tm->d_p1[i].z;
	device_cross(&cm->d_q.x[i], &cm->d_q.y[i],&cm->d_q.z[i], cm->d_t.x[i], cm->d_t.y[i], cm->d_t.z[i], tm->d_edges.e1.x[i], tm->d_edges.e1.y[i], tm->d_edges.e1.z[i]);
	cm->d_w[i] = device_dot(cm->d_q.x[i], cm->d_q.y[i], cm->d_q.z[i], tm->d_edges.e2.x[i], tm->d_edges.e2.y[i], tm->d_edges.e2.z[i]);
}

__global__ void intersect_voxel_cuda( Camera::pixel_memory* cm,  Camera::voxel_memory* cvm, Camera::trixel_memory* ctm, Trixel::trixel_memory* tm, s64 max_threads) {
	s64 i = (s64)threadIdx.x + ((s64)blockIdx.x * blockDim.x);
	if (i >= max_threads) { return; }

	double d = 400.0;//**TODO** get rid of hardcode
	double t0x, t1x, t0y, t1y, t0z, t1z, maxt0, mint1, dir, s1, s2;
	s64 cni, index_front = i * cvm->index_queue_offset, index_start = i * cvm->index_queue_offset;
	s64 tri_i;
	double pe1, f, p_x, p_y, p_z, u, v, w;
	cvm->d_voxel_index_queue[index_front] = 0;
	//cm->d_color.rad[i].r = 0;
	//cm->d_color.rad[i].g = 0;
	//cm->d_color.rad[i].b = 0;
	cm->d_rmi.index[i] = (s64)-1;
	cm->d_dist.d[i] = d;

	while ((index_front - index_start) >= 0) {
		cni = cvm->d_voxel_index_queue[index_front--];
		if (cvm->is_leaf[cni]) {
			tri_i = cvm->children[cni].triangle;
			device_cross(&p_x, &p_y, &p_z,
				cm->rmd.d_x[i], cm->rmd.d_y[i], cm->rmd.d_z[i],
				tm->d_edges.e2.x[tri_i], tm->d_edges.e2.y[tri_i], tm->d_edges.e2.z[tri_i]);
			f = device_dot(p_x, p_y, p_z,
				tm->d_edges.e1.x[tri_i], tm->d_edges.e1.y[tri_i], tm->d_edges.e1.z[tri_i]);
			if (!(f < MOLLER_TRUMBORE_DEVICE_EPSILON && f > -MOLLER_TRUMBORE_DEVICE_EPSILON)) {
				pe1 = 1.0 / f;
				u = pe1 * device_dot(p_x, p_y, p_z,
					ctm->d_t.x[tri_i], ctm->d_t.y[tri_i], ctm->d_t.z[tri_i]);
				v = pe1 * device_dot(cm->rmd.d_x[i], cm->rmd.d_y[i], cm->rmd.d_z[i],
					ctm->d_q.x[tri_i], ctm->d_q.y[tri_i], ctm->d_q.z[tri_i]);
				w = pe1 * ctm->d_w[tri_i];
				if ((w < d) && !((u < MOLLER_TRUMBORE_DEVICE_EPSILON) || (v < MOLLER_TRUMBORE_DEVICE_EPSILON) || ((u + v) > 1 + MOLLER_TRUMBORE_DEVICE_EPSILON) || (w < MOLLER_TRUMBORE_DEVICE_EPSILON))) {
					//cm->d_rmi.index[i] = -2;
					//return;
					cm->d_rmi.index[i] = tri_i;
					cm->d_dist.d[i] = w;
					cm->d_color.rad[i].r = tm->d_color.rad[tri_i].r;
					cm->d_color.rad[i].g = tm->d_color.rad[tri_i].g;
					cm->d_color.rad[i].b = tm->d_color.rad[tri_i].b;
					cm->pnt.d_x[i] = d * cm->rmd.d_x[i];
					cm->pnt.d_y[i] = d * cm->rmd.d_y[i];
					cm->pnt.d_z[i] = d * cm->rmd.d_z[i];
					cm->norm.d_x[i] = tm->d_n.x[tri_i];
					cm->norm.d_y[i] = tm->d_n.y[tri_i];
					cm->norm.d_z[i] = tm->d_n.z[tri_i];
					return;
				}
			}	
			continue;
		}
		//YAY NO BRANCHES  ???? is it worth??? NO IDEA
			//swap t0, t1 if ray in negative direction
			//**TODO** precomput sign * rmd to remove more operations
		t0x = ((cvm->d_Bo[cni].t0x * (1 - cm->sign_rmd.d_x[i])) + (cvm->d_Bo[cni].t1x * (cm->sign_rmd.d_x[i]))) * cm->inv_rmd.d_x[i];
		t1x = ((cvm->d_Bo[cni].t1x * (1 - cm->sign_rmd.d_x[i])) + (cvm->d_Bo[cni].t0x * (cm->sign_rmd.d_x[i]))) * cm->inv_rmd.d_x[i];

		t0y = ((cvm->d_Bo[cni].t0y * (1 - cm->sign_rmd.d_y[i])) + (cvm->d_Bo[cni].t1y * (cm->sign_rmd.d_y[i]))) * cm->inv_rmd.d_y[i];
		t1y = ((cvm->d_Bo[cni].t1y * (1 - cm->sign_rmd.d_y[i])) + (cvm->d_Bo[cni].t0y * (cm->sign_rmd.d_y[i]))) * cm->inv_rmd.d_y[i];

		t0z = ((cvm->d_Bo[cni].t0z * (1 - cm->sign_rmd.d_z[i])) + (cvm->d_Bo[cni].t1z * (cm->sign_rmd.d_z[i]))) * cm->inv_rmd.d_z[i];
		t1z = ((cvm->d_Bo[cni].t1z * (1 - cm->sign_rmd.d_z[i])) + (cvm->d_Bo[cni].t0z * (cm->sign_rmd.d_z[i]))) * cm->inv_rmd.d_z[i];
		//select entrance (maxt0) and exit(mint1) planes of voxel, and then get coordinate in split direction ( t * dir)
		//if ((t0x > t1y) || (t0y > t1x)) { return; }
		maxt0 = fmax(t0z, fmax(t0x, t0y));
		mint1 = fmin(t1z, fmin(t1x, t1y));
		//if ((maxt0 > t1z) || (t0z > mint1)) { return; }

		if (mint1 > maxt0 - DEVICE_EPSILON_SINGLE && maxt0 > -DEVICE_EPSILON_SINGLE) {
			//cm->d_rmi.index[i] = (s64)-2;
			dir = ((cm->rmd.d_x[i] * cvm->cut_flags[cni].x) + (cm->rmd.d_y[i] * cvm->cut_flags[cni].y) + (cm->rmd.d_z[i] * cvm->cut_flags[cni].z));
			maxt0 *= dir; mint1 *= dir;

			s1 = cvm->s1[cni] + DEVICE_EPSILON_SINGLE;
			s2 = cvm->s2[cni] ;
			if (maxt0  < s2 + DEVICE_EPSILON_SINGLE) {
				//cm->d_color.rad[i].r += .001;

				if (mint1 > s2 - DEVICE_EPSILON_SINGLE) { 
					cvm->d_voxel_index_queue[++index_front] = cvm->children[cni].right;
					//cm->d_color.rad[i].r += .0005;

				}
				cvm->d_voxel_index_queue[++index_front] = cvm->children[cni].left;
			}
			else {
				//cm->d_color.rad[i].b += .001;
				if (mint1  < s1 || maxt0 < s1 ) {
					cvm->d_voxel_index_queue[++index_front] = cvm->children[cni].left;
					//cm->d_color.rad[i].r += .0005;

				}
				cvm->d_voxel_index_queue[++index_front] = cvm->children[cni].right;
			}
		}
	}
	return;
}
__global__ void intersect_tri_cuda(Trixel::trixel_memory* tm, Camera::pixel_memory* cm, Camera::trixel_memory* ctm,
	s64 num_trixels, u64 max_threads) {
	u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
	if (i >= max_threads) { return; }
	s64 tri_index = -1;
	double d = 400.0;//**TODO** get rid of hardcode
	double pe1,f, p_x, p_y, p_z, u, v, w;
	//DO moller TUMBORE ON EACH TRIANGLE, set index to closest one.
	for (int tri_i = 0; tri_i < num_trixels; tri_i++) {
		device_cross(&p_x, &p_y, &p_z,
			cm->rmd.d_x[i], cm->rmd.d_y[i], cm->rmd.d_z[i],
			tm->d_edges.e2.x[tri_i], tm->d_edges.e2.y[tri_i], tm->d_edges.e2.z[tri_i]);
		f = device_dot(p_x, p_y, p_z,
			tm->d_edges.e1.x[tri_i], tm->d_edges.e1.y[tri_i], tm->d_edges.e1.z[tri_i]);
		if (!(f < DEVICE_EPSILON_SINGLE && f > -DEVICE_EPSILON_SINGLE)) {
			pe1 = 1.0 / f;
			u = pe1 * device_dot(p_x, p_y, p_z,
				ctm->d_t.x[tri_i], ctm->d_t.y[tri_i], ctm->d_t.z[tri_i]);
			v = pe1 * device_dot(cm->rmd.d_x[i], cm->rmd.d_y[i], cm->rmd.d_z[i],
				ctm->d_q.x[tri_i], ctm->d_q.y[tri_i], ctm->d_q.z[tri_i]);
			w = pe1 * ctm->d_w[tri_i];
			if ((w < d) && !((u < DEVICE_EPSILON_SINGLE) || (v < DEVICE_EPSILON_SINGLE) || ((u + v) > 1) || (w < DEVICE_EPSILON_SINGLE))) {
				d = w; tri_index = tri_i;
			}
		}
	}
	cm->d_rmi.index[i] = tri_index;
	cm->d_dist.d[i] = d;
	if (tri_index != (s64)-1) {
		cm->pnt.d_x[i] = d * cm->rmd.d_x[i];
		cm->pnt.d_y[i] = d * cm->rmd.d_y[i];
		cm->pnt.d_z[i] = d * cm->rmd.d_z[i];
		cm->norm.d_x[i] = tm->d_n.x[tri_index];
		cm->norm.d_y[i] = tm->d_n.y[tri_index];
		cm->norm.d_z[i] = tm->d_n.z[tri_index];
	}
}
hipError_t intersect_trixels_device(Trixel* t, Camera* c, u32 mode) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}

	if (mode == 0) {
		intersect_voxel_cuda << < 1 + (u32)(c->f_prop.res.count / 128), 128 >> > (
			(Camera::pixel_memory*)c->d_mem,
			(Camera::voxel_memory*)c->d_voxels,
			(Camera::trixel_memory*)c->d_trixels,
			(Trixel::trixel_memory*)t->d_mem,
			c->f_prop.res.count);		
	}
	else {
		intersect_tri_cuda << < 1 + (u32)(c->f_prop.res.count / BLOCK_SIZE), BLOCK_SIZE >> > (
			(Trixel::trixel_memory*)t->d_mem,
			(Camera::pixel_memory*)c->d_mem,
			(Camera::trixel_memory*)c->d_trixels,
			t->num_trixels, c->f_prop.res.count);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("intersect_trixels_device launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching intersect_trixels_device!\n", cudaStatus);
	}


	return cudaStatus;

}

hipError_t init_camera_trixel_device_memory(Trixel* t, Camera* c) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	init_cam_tri_mem_cuda << < 1 + (u32)(t->num_trixels / BLOCK_SIZE), BLOCK_SIZE >> > ((Trixel::trixel_memory*)t->d_mem, (Camera::trixel_memory*)c->d_trixels,  c->o_prop.pos, t->num_trixels);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("init_camera_trixel_device_memory launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching init_camera_trixel_device_memory!\n", cudaStatus);
	}
	return cudaStatus;
}

hipError_t init_trixels_device_memory(Trixel* t) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	init_tri_mem_cuda << < 1 + (u32)(t->num_trixels / BLOCK_SIZE), BLOCK_SIZE >> > ((Trixel::trixel_memory*)t->d_mem, t->d_points_init_data, t->num_trixels);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("init_trixels_device_memory launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching init_tri_mem_cuda!\n", cudaStatus);
	}
	hipMemcpy(t->h_mem.h_p1, t->h_mem.d_p1, t->num_trixels * sizeof(double), hipMemcpyDeviceToHost);

	return cudaStatus;
}

