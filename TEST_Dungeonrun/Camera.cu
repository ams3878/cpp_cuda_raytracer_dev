#include "hip/hip_runtime.h"
﻿#ifndef CUDA_KERNEL
#define CUDA_KERNEL
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#endif
#include "platform_common.h"
#include "Camera.h"
#include "Trixel.h"
#ifndef CUDA_VECTOR_H
#include "vector.cuh" 
#endif

__global__ void color_cam_cuda(Camera::pixel_memory* cm, u32* t, u64 max_threads) {
    u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
    if (i >= max_threads) { return; }
    u32 pattern = i % 16;
    s64 tri_index = cm->d_rmi.index[i];
    if (tri_index != -1) {
        cm->d_color.c[i] = t[tri_index];
    }
    else {
        cm->d_color.argb[i].r = (u8)240;  cm->d_color.argb[i].b = (u8)0; cm->d_color.argb[i].g = (u8)0; cm->d_color.argb[i].a = (u8)0; 
         if (pattern < 8) { cm->d_color.argb[i].g = (u8)240; cm->d_color.argb[i].r = (u8)0; }        
    }
}
hipError_t color_camera_device(Camera* c) {
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    color_cam_cuda << < 1 + ((u32)c->f_prop.res.count / BLOCK_SIZE), BLOCK_SIZE >> > ((Camera::pixel_memory*)c->d_mem, c->trixels_list->h_mem.d_color.c, c->f_prop.res.count);
    status_lauch_and_sync(color_cam_cuda);
    hipMemcpy(c->h_mem.h_color.c, c->h_mem.d_color.c, c->f_prop.res.count * sizeof(u32), hipMemcpyDeviceToHost);

    return cudaStatus;
}


__global__ void init_cam_mem_cuda(Camera::pixel_memory* m, u64 res_w, u64 res_h, double draw_distance, double pix_w, double pix_h,
    Camera::orientation_properties::o_vector n,    Camera::orientation_properties::o_vector v,    Camera::orientation_properties::o_vector u){
    //MAYBE make o_vectors shared mem? if you do delete the o_vector mod memebers
    u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
    if (i >= (res_w * res_h)) { return; }
    u64 i_y = i / res_w;
    u64 i_x = i % res_w;
    u32 pattern = i % 8;

    m->rad.d_r[i] = 0.0;    m->rad.d_g[i] = 0.0;    m->rad.d_b[i] = 0.0;
    m->d_color.argb[i].r = (u8)240;  m->d_color.argb[i].b = (u8)0; m->d_color.argb[i].g = (u8)0; m->d_color.argb[i].a = (u8)0;
    if(pattern < 4){ m->d_color.argb[i].g = (u8)240; m->d_color.argb[i].r = (u8)0;}
    m->norm.d_x[i] = 0.0;   m->norm.d_y[i] = 0.0;   m->norm.d_z[i] = 0.0;
    m->pnt.d_x[i] = 0.0;    m->pnt.d_y[i] = 0.0;    m->pnt.d_z[i] = 0.0;


    m->rmd.d_x[i] = n.x + u.x * i_x + v.x * i_y;    m->rmd.d_y[i] = n.y + u.y * i_x + v.y * i_y;    m->rmd.d_z[i] = n.z + u.z * i_x + v.z * i_y;
    device_normalize_vector(&m->rmd.d_x[i], &m->rmd.d_y[i], &m->rmd.d_z[i]);

    m->inv_rmd.d_x[i] = 1/m->rmd.d_x[i];    m->inv_rmd.d_y[i] = 1/ m->rmd.d_y[i];    m->inv_rmd.d_z[i] = 1/ m->rmd.d_z[i];
    m->sign_rmd.d_x[i] = ((u64*)m->rmd.d_x)[i] >> 63;    m->sign_rmd.d_y[i] = ((u64*)m->rmd.d_y)[i] >> 63;    m->sign_rmd.d_z[i] = ((u64*)m->rmd.d_z)[i] >> 63;

    m->d_dist.d[i] = draw_distance;
    m->d_rmi.index[i] = -1;    
}

hipError_t init_camera_device_memory(Camera* c){
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }  
    init_cam_mem_cuda << < 1+((u32)c->f_prop.res.count / BLOCK_SIZE), BLOCK_SIZE >>> ((Camera::pixel_memory*)c->d_mem, c->f_prop.res.w, c->f_prop.res.h, c->r_prop.draw_distance,
        c->f_prop.pix.w, c->f_prop.pix.h, c->o_prop.n_mod, c->o_prop.v_mod, c->o_prop.u_mod);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("init_cam_mem_cuda launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize returned error code %d after launching init_cam_mem_cuda!\n", cudaStatus);
    }
    hipMemcpy(c->h_mem.h_rmi.index, c->h_mem.d_rmi.index, c->f_prop.res.count * sizeof(u64), hipMemcpyDeviceToHost);
    hipMemcpy(c->h_mem.h_color.c, c->h_mem.d_color.c, c->f_prop.res.count * sizeof(u32), hipMemcpyDeviceToHost);
    return cudaStatus;
}

__global__ void init_cam_voxel_mem_cuda(Camera::voxel_memory* vm, Trixel::kd_tree* kdm, vector_xyz co, u64 max_threads) {

    u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
    if (i >= max_threads) { return; }
    int cut_dir = kdm->d_nodes[i].cut_flag;
    vm->d_Bo[i].t0x = kdm->d_nodes[i].h_bound.x0 - co.x;
    vm->d_Bo[i].t1x = kdm->d_nodes[i].h_bound.x1 - co.x;
    vm->d_Bo[i].t0y = kdm->d_nodes[i].h_bound.y0 - co.y;
    vm->d_Bo[i].t1y = kdm->d_nodes[i].h_bound.y1 - co.y;
    vm->d_Bo[i].t0z = kdm->d_nodes[i].h_bound.z0 - co.z;
    vm->d_Bo[i].t1z = kdm->d_nodes[i].h_bound.z1 - co.z;
    vm->is_leaf[i] = kdm->d_nodes[i].is_leaf;
    vm->children[i].left = kdm->d_nodes[i].left_node;
    vm->children[i].right= kdm->d_nodes[i].right_node;
    vm->children[i].triangle = vm->is_leaf[i] == 0 ? -1 : kdm->d_nodes[i].tri_index;

    vm->cut_flags[i].x = cut_dir == 0 || cut_dir == 3 ? 1 : 0;
    vm->cut_flags[i].y = cut_dir == 1 || cut_dir == 4 ? 1 : 0;
    vm->cut_flags[i].z = cut_dir == 2 || cut_dir == 5 ? 1 : 0;


    vm->s1[i] = kdm->d_nodes[i].s1 - ((co.x * (double)vm->cut_flags[i].x) + (co.y * (double)vm->cut_flags[i].y) + (co.z * (double)vm->cut_flags[i].z));
    vm->s2[i] = kdm->d_nodes[i].s2 - ((co.x * (double)vm->cut_flags[i].x) + (co.y * (double)vm->cut_flags[i].y) + (co.z * (double)vm->cut_flags[i].z));

}

hipError_t init_camera_voxel_device_memory(Trixel* t, Camera* c) {
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    init_cam_voxel_mem_cuda << < 1 + (u32)(t->num_voxels / BLOCK_SIZE)  , BLOCK_SIZE >> > (
        (Camera::voxel_memory*)c->d_voxels,
        (Trixel::kd_tree*)t->d_tree,
        c->o_prop.pos, t->num_voxels);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("init_cam_voxel_cuda launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize returned error code %d after launching init_cam_mem_cuda!\n", cudaStatus);
    }
    return cudaStatus;
}

__global__ void update_cam_voxel_mem_cuda(Camera::voxel_memory* vm, Input::translate_vector tv, u64 max_threads) {

    u64 i = (u64)threadIdx.x + ((u64)blockIdx.x * blockDim.x);
    if (i >= max_threads) { return; }
    //THESE VALUES ARE ALL CALCULATED AS SOME POINT IN SPACE minus THE CAMERA ORIGIN
    //So if you move the camera by  + dx...need to subtract dx from the voxel
    vm->d_Bo[i].t0x -= tv.dx;
    vm->d_Bo[i].t1x -= tv.dx;
    vm->d_Bo[i].t0y -= tv.dy;
    vm->d_Bo[i].t1y -= tv.dy;
    vm->d_Bo[i].t0z -= tv.dz;
    vm->d_Bo[i].t1z -= tv.dz;

    vm->s1[i] -= (tv.dx * (double)vm->cut_flags[i].x) + (tv.dy * (double)vm->cut_flags[i].y) + (tv.dz * (double)vm->cut_flags[i].z);
    vm->s2[i] -= (tv.dx * (double)vm->cut_flags[i].x) + (tv.dy * (double)vm->cut_flags[i].y) + (tv.dz * (double)vm->cut_flags[i].z);

}
hipError_t update_camera_voxel_device_memory(Input::translate_vector tv, Camera* c) {
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    update_cam_voxel_mem_cuda << < 1 + (u32)(c->h_voxels.num_voxels / BLOCK_SIZE), BLOCK_SIZE >> > (
        (Camera::voxel_memory*)c->d_voxels,
        tv, c->h_voxels.num_voxels);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("init_cam_voxel_cuda launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize returned error code %d after launching init_cam_mem_cuda!\n", cudaStatus);
    }
    return cudaStatus;
}

